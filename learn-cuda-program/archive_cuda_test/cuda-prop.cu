#include <iostream>
#include <hip/hip_runtime.h>
#include <cstdio>
#define STR_HELPER(x) #x
#define STR(x) STR_HELPER(x)

__device__ void print_arch(){
  const char my_compile_time_arch[] = STR(__CUDA_ARCH__);
  printf("__CUDA_ARCH__: %s\n", my_compile_time_arch);
}
//nvcc cuda-prop.cu -arch=sm_89 -o cuda_prop否则 __CUDA_ARCH__ = 520

__global__ void example()
{
   print_arch();
}

int main(void){
    int dev = 0;
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, dev);
    std::cout << "使用GPU device " << dev << ": " << devProp.name << std::endl;
    std::cout << "SM的数量：" << devProp.multiProcessorCount << std::endl;
    std::cout << "每个线程块的共享内存大小：" << devProp.sharedMemPerBlock / 1024.0 << " KB" << std::endl;
    std::cout << "每个线程块的最大线程数：" << devProp.maxThreadsPerBlock << std::endl;
    std::cout << "每个EM的最大线程数：" << devProp.maxThreadsPerMultiProcessor << std::endl;
    std::cout << "每个SM的最大线程束数：" << devProp.maxThreadsPerMultiProcessor / 32 << std::endl;

    example<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}

