#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<iostream>
#include<sys/time.h>

struct Matrix
{
    int width;
    int height;
    float *elements;
};

// 获取矩阵A的(row, col)元素
__device__ float getElement(Matrix *A, int row, int col)
{
	return A->elements[row * A->width + col];
}

// 为矩阵A的(row, col)元素赋值
__device__ void setElement(Matrix *A, int row, int col, float value)
{
	A->elements[row * A->width + col] = value;
}

// 矩阵相乘kernel，2-D，每个线程计算一个元素
__global__ void matMulKernel(Matrix *A, Matrix *B, Matrix *C)
{
	float Cvalue = 0.0;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	for (int i = 0; i < A->width; ++i)
	{
		Cvalue += getElement(A, row, i) * getElement(B, i, col);
	}
	setElement(C, row, col, Cvalue);
}

int main()
{
    int width = 1 << 10;
    int height = 1 << 10;
    Matrix *A, *B, *C;
    // 申请托管内存
    hipMallocManaged((void**)&A, sizeof(Matrix));
    hipMallocManaged((void**)&B, sizeof(Matrix));
    hipMallocManaged((void**)&C, sizeof(Matrix));
    int nBytes = width * height * sizeof(float);
    hipMallocManaged((void**)&A->elements, nBytes);
    hipMallocManaged((void**)&B->elements, nBytes);
    hipMallocManaged((void**)&C->elements, nBytes);

    // 初始化数据
    A->height = height;
    A->width = width;
    B->height = height;
    B->width = width;
    C->height = height;
    C->width = width;
    for (int i = 0; i < width * height; ++i)
    {
        A->elements[i] = 1.0;
        B->elements[i] = 2.0;
    }

    // 定义kernel的执行配置
    dim3 blockSize(32, 32);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, 
        (height + blockSize.y - 1) / blockSize.y);


    struct timeval t1,t2;
    // 执行kernel
    gettimeofday(&t1,NULL);
    matMulKernel << < gridSize, blockSize >> >(A, B, C);
    hipDeviceSynchronize();
    gettimeofday(&t2,NULL);
    
    float time_use = (t2.tv_sec - t1.tv_sec) * 1000 + (t2.tv_usec - t1.tv_usec) / 1000;
    std::cout<<"Time use: "<<time_use<<" ms"<<std::endl;

    // 检查执行结果
    float maxError = 0.0;
    for (int i = 0; i < width * height; ++i)
        maxError = fmax(maxError, fabs(C->elements[i] - 2 * width));
    std::cout << "最大误差: " << maxError << std::endl;

    return 0;
}